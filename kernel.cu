
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32  // also the warpSize
#define BLOCK_SIZE (N * N)

__inline__ __device__
int warpSum(int val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        val += __shfl_down(val, offset);
    }
    return val;
}

__inline__ __device__
int blockSum(int val) {
    static __shared__ int shared[N]; // shared mem for 32 partial sums
    int x = threadIdx.x % warpSize;
    int y = threadIdx.x / warpSize;

    val = warpSum(val);              // reduce warp
    if (x == 0) shared[y] = val;     // write result to shared memory
    __syncthreads();

    val = shared[x];
    if (y == 0) val = warpSum(val);  // reduce within 1st warp

    return val;
}

__global__
void reduceRows(const int *in, float *out, int X, int Y) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int sum = 0;
    for (int i = x; i < X*Y; i += blockDim.x * gridDim.x) {
        sum += in[i];
    }
    sum = blockSum(sum);
    if (threadIdx.x == 0) {
        atomicAdd(out + x / X, sum);
    }
}

__global__
void reduceCols(const int *in, float *out, int X, int Y) {
    int y = blockIdx.x*blockDim.x + threadIdx.x;
    int row = y % X;
    int col = y / X;
    int sum = in[col*X + row];
    // for (int i = y; i < X*Y; i += blockDim.x * gridDim.x) {
    //     sum += in[i*X + col];
    // }
    sum = blockSum(sum);
    if (threadIdx.x == 0) {
        atomicAdd(out + col, sum);
    }
}

__global__
void nullify(float *arr) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    arr[i] = 0;
}

__global__
void divide(float *arr, float count) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    arr[i] /= count;
}

void solveGPU(
    const int *results,  // questions * students
    float *avg_stud,     // score per student: total / questions -> len Y
    float *avg_que,      // score per question: total / students -> len X
    const int Y,         // students: always divisible by 32
    const int X          // questions: always divisible by 32
) {
    int n = X * Y;

    // reset arrays
    nullify<<<Y/N, N>>>(avg_stud);
    nullify<<<X/N, N>>>(avg_que);

    dim3 threads(BLOCK_SIZE);
    dim3 blocks(n/BLOCK_SIZE);

    // load all results
    reduceCols<<<blocks, threads>>>(results, avg_que, X, Y);
    reduceRows<<<blocks, threads>>>(results, avg_stud, X, Y);

    // divide results
    divide<<<Y/N, N>>>(avg_stud, X);
    divide<<<X/N, N>>>(avg_que, Y);

    if (hipPeekAtLastError() != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(hipGetLastError()));
    }
}
