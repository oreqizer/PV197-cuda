/*
TODO:
- reduce 'y' block size by 2 or 4
- switch reading to 'int2' or 'int4'
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32  // also the tile size
#define BLOCK_SIZE (N * N)

__inline__ __device__
int warpSum(int val) {
    // warpSize always 32
    val += __shfl_down(val, 16);
    val += __shfl_down(val, 8);
    val += __shfl_down(val, 4);
    val += __shfl_down(val, 2);
    val += __shfl_down(val, 1);
    return val;
}

__global__
void reduce(const int *in, float *out_stud, float *out_que, int X, int Y) {
    __shared__ int tile[N][N + 1];      // bank conflict
    int x = blockIdx.x*N + threadIdx.x;
    int y = blockIdx.y*N + threadIdx.y;
    int width = gridDim.x*N;            // width of the whole matrix
    int idx = y*width + x;              // global index
    int val = in[idx];

    tile[threadIdx.x][threadIdx.y] = val;
    __syncthreads();

    register int sum_stud = val;
    register int sum_que = tile[threadIdx.y][threadIdx.x];

    sum_stud = warpSum(sum_stud);
    sum_que = warpSum(sum_que);

    if (threadIdx.x == 0) {
        int stud_i = idx / X;
        int que_i = y + (idx & X - 1);
        atomicAdd(out_stud + stud_i, sum_stud);
        atomicAdd(out_que + que_i, sum_que);
    }
}

__global__
void divide(float *arr, float count) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    arr[i] /= count;
}

void solveGPU(
    const int *results,  // questions * students
    float *avg_stud,     // score per student: total / questions -> len Y
    float *avg_que,      // score per question: total / students -> len X
    const int Y,         // students: always divisible by 32
    const int X          // questions: always divisible by 32
) {
    int n = X * Y;

    // reset arrays
    hipMemset(avg_stud, 0, Y*sizeof(avg_stud[0]));
    hipMemset(avg_que, 0, X*sizeof(avg_que[0]));

    dim3 threads(N, N);
    dim3 blocks(n/BLOCK_SIZE);

    // load all results
    reduce<<<blocks, threads>>>(results, avg_stud, avg_que, X, Y);

    // divide results - TODO merge to 'reduce'
    divide<<<Y/N, N>>>(avg_stud, X);
    divide<<<X/N, N>>>(avg_que, Y);

    if (hipPeekAtLastError() != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(hipGetLastError()));
    }
}
