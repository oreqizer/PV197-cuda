
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void solver(
    const int *results,
    float *avg_stud,
    float *avg_que,
    const int dim_x,  // always divisible by 32
    const int dim_y  // always divisible by 32
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int resultIds = y*dim_x + x;

    // add to resulting array
    avg_que[x] += results[resultIds];
    avg_stud[y] += results[resultIds];
}

void solveGPU(
    const int *results,  // students * questions
    float *avg_stud,
    float *avg_que,
    const int students,  // y: always divisible by 32
    const int questions  // x: always divisible by 32
) {
    solver<<<questions, students>>>(
        results, avg_stud, avg_que, questions, students
    );
}
